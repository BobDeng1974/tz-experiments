#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

__global__ void _ccv_nnc_categorical_crossentropy_forw_kernel(const int batch_size, const int count, const float* const label, const float* const a, float* const c)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const int idx = (int)(label[i] + 0.5);
		c[i] = -logf(a[i * count + idx]);
	}
}

__global__ void _ccv_nnc_categorical_crossentropy_forw_kernel(const int batch_size, const int count, const int* const label, const float* const a, float* const c)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		c[i] = -logf(a[i * count + label[i]]);
	}
}

static int _ccv_nnc_categorical_crossentropy_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, const ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 2);
	const ccv_nnc_tensor_t* a = inputs[0];
	assert(!CCV_IS_TENSOR_VIEW(a));
	const ccv_nnc_tensor_t* b = inputs[1];
	assert(!CCV_IS_TENSOR_VIEW(b));
	assert(output_size == 1);
	ccv_nnc_tensor_t* c = outputs[0];
	assert(!CCV_IS_TENSOR_VIEW(c));
	const int axis_count = ccv_nnc_tensor_nd(a->info.dim);
	const int batch_size = axis_count < 2 ? 1 : a->info.dim[0];
	const int count = ccv_nnc_tensor_count(a->info) / batch_size;
	int i;
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	const int device = ccv_nnc_stream_context_get_device(stream_context);
	hipSetDevice(device);
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && b->info.dim[i] > 0; i++)
		{ assert(b->info.dim[i] == c->info.dim[i]); }
	if (b->info.datatype == CCV_32F)
		_ccv_nnc_categorical_crossentropy_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.f32, a->data.f32, c->data.f32);
	else if (b->info.datatype == CCV_32S)
		_ccv_nnc_categorical_crossentropy_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.i32, a->data.f32, c->data.f32);
	return CCV_NNC_EXEC_SUCCESS;
}

__global__ void _ccv_nnc_set_zero_kernel(const int n, float* const a)
{
	CUDA_1D_KERNEL_LOOP(i, n) {
		a[i] = 0;
	}
}

__global__ void _ccv_nnc_categorical_crossentropy_back_kernel(const int batch_size, const int count, const float* const g, const float* const label, const float* const a, float* const h)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const int idx = (int)(label[i] + 0.5);
		h[i * count + idx] = -g[i] / a[i * count + idx];
	}
}

__global__ void _ccv_nnc_categorical_crossentropy_back_kernel(const int batch_size, const int count, const float* const g, const int* const label, const float* const a, float* const h)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const int idx = label[i];
		h[i * count + idx] = -g[i] / a[i * count + idx];
	}
}

__global__ void _ccv_nnc_categorical_crossentropy_back_kernel(const int batch_size, const int count, const float* const label, const float* const a, float* const h)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const int idx = (int)(label[i] + 0.5);
		h[i * count + idx] = -1. / a[i * count + idx];
	}
}

__global__ void _ccv_nnc_categorical_crossentropy_back_kernel(const int batch_size, const int count, const int* const label, const float* const a, float* const h)
{
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const int idx = label[i];
		h[i * count + idx] = -1. / a[i * count + idx];
	}
}

static int _ccv_nnc_categorical_crossentropy_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, const ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 3);
	assert(output_size >= 1);
	const ccv_nnc_tensor_t* g = inputs[0];
	assert(!g || !CCV_IS_TENSOR_VIEW(g));
	const ccv_nnc_tensor_t* a = inputs[1];
	assert(!CCV_IS_TENSOR_VIEW(a));
	const ccv_nnc_tensor_t* b = inputs[2];
	assert(!CCV_IS_TENSOR_VIEW(b));
	ccv_nnc_tensor_t* h = outputs[0];
	assert(!CCV_IS_TENSOR_VIEW(h));
	const int axis_count = ccv_nnc_tensor_nd(a->info.dim);
	const int batch_size = axis_count < 2 ? 1 : a->info.dim[0];
	const int bcount = ccv_nnc_tensor_count(a->info);
	const int count = bcount / batch_size;
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
		{ assert(a->info.dim[i] == h->info.dim[i]); }
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	const int device = ccv_nnc_stream_context_get_device(stream_context);
	hipSetDevice(device);
	_ccv_nnc_set_zero_kernel<<<CUDA_GET_BLOCKS(bcount), CUDA_NUM_THREADS, 0, stream>>>(bcount, h->data.f32);
	if (g)
	{
		if (b->info.datatype == CCV_32F)
			_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, g->data.f32, b->data.f32, a->data.f32, h->data.f32);
		else if (b->info.datatype == CCV_32S)
			_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, g->data.f32, b->data.i32, a->data.f32, h->data.f32);
	} else {
		if (b->info.datatype == CCV_32F)
			_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.f32, a->data.f32, h->data.f32);
		else if (b->info.datatype == CCV_32S)
			_ccv_nnc_categorical_crossentropy_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, b->data.i32, a->data.f32, h->data.f32);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_CATEGORICAL_CROSSENTROPY_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_32S;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_categorical_crossentropy_forw;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_CATEGORICAL_CROSSENTROPY_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_32S;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_categorical_crossentropy_back;
}
