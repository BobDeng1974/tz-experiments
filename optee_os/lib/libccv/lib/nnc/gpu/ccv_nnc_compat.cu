#include "hip/hip_runtime.h"
#include "ccv_nnc_compat.h"
extern "C" {
#include <nnc/ccv_nnc_easy.h>
}

void* cumalloc(int device, size_t size)
{
	void* ptr = 0;
	hipSetDevice(device);
	hipMalloc(&ptr, size);
	return ptr;
}

void cufree(int device, void* ptr)
{
	hipSetDevice(device);
	hipFree(ptr);
}

typedef struct {
	int type; // Kept the type specifier.
	hipStream_t stream;
	hipblasHandle_t cublas;
	struct {
		int n;
		float* data;
	} ones;
#ifdef HAVE_CUDNN
	hipdnnHandle_t cudnn;
	void* rngs; // user-allocated GPU memory that will hold random number generator states.
	unsigned long long seed;
#endif
} ccv_nnc_stream_context_compat_t;

static __thread ccv_nnc_stream_context_compat_t ccv_nnc_per_thread_gpu_stream_context = {
	.type = CCV_STREAM_CONTEXT_GPU,
#ifdef hipStreamPerThread
	.stream = hipStreamPerThread
#endif
};

ccv_nnc_stream_context_t* ccv_nnc_init_stream_context(ccv_nnc_stream_context_t* const stream_context)
{
	assert(CCV_STREAM_GET_CONTEXT(((int*)stream_context)[0]) == CCV_STREAM_CONTEXT_GPU);
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)ccrealloc(stream_context, sizeof(ccv_nnc_stream_context_compat_t));
	int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
	hipSetDevice(device);
	hipStreamCreate(&stream_compat->stream);
	stream_compat->cublas = 0;
	stream_compat->ones.data = 0;
#ifdef HAVE_CUDNN
	stream_compat->cudnn = 0;
	stream_compat->rngs = 0;
#endif
	return (ccv_nnc_stream_context_t*)stream_compat;
}

void ccv_nnc_synchronize_stream_context(const ccv_nnc_stream_context_t* const stream_context)
{
	const ccv_nnc_stream_context_compat_t* stream_compat = (const ccv_nnc_stream_context_compat_t*)stream_context;
	int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
	hipSetDevice(device);
	hipStreamSynchronize(stream_compat->stream);
}

void ccv_nnc_deinit_stream_context(ccv_nnc_stream_context_t* const stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
	hipSetDevice(device);
	hipStreamDestroy(stream_compat->stream);
	if (stream_compat->cublas)
		hipblasDestroy(stream_compat->cublas);
	if (stream_compat->ones.data)
		hipFreeAsync(stream_compat->ones.data, stream_compat->stream);
#ifdef HAVE_CUDNN
	if (stream_compat->cudnn)
		hipdnnDestroy(stream_compat->cudnn);
	if (stream_compat->rngs)
		hipFreeAsync(stream_compat->rngs, stream_compat->stream);
#endif
}

int ccv_nnc_stream_context_get_device(const ccv_nnc_stream_context_t* const stream_context)
{
	if (!stream_context)
		return 0;
	const ccv_nnc_stream_context_compat_t* stream_compat = (const ccv_nnc_stream_context_compat_t*)stream_context;
	return CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
}

hipStream_t ccv_nnc_stream_context_get_stream(const ccv_nnc_stream_context_t* const stream_context)
{
	const ccv_nnc_stream_context_compat_t* stream_compat = (const ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = &ccv_nnc_per_thread_gpu_stream_context;
	return stream_compat->stream;
}

hipblasHandle_t ccv_nnc_stream_context_get_cublas(const ccv_nnc_stream_context_t* const stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = &ccv_nnc_per_thread_gpu_stream_context;
	if (!stream_compat->cublas)
	{
		int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
		hipSetDevice(device);
		hipblasCreate(&stream_compat->cublas);
		hipblasSetStream(stream_compat->cublas, stream_compat->stream);
	}
	return stream_compat->cublas;
}

// A simple kernel to set all values to 1.
__global__ static void ones(float* x, int n)
{
	const int thidx = blockIdx.x * blockDim.x + threadIdx.x;
	if (thidx < n)
		x[thidx] = 1.;
}

float* ccv_nnc_stream_context_get_ones(const ccv_nnc_stream_context_t* const stream_context, const int n)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = &ccv_nnc_per_thread_gpu_stream_context;
	if (!stream_compat->ones.data || n > stream_compat->ones.n)
	{
		int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
		hipSetDevice(device);
		hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
		if (stream_compat->ones.data)
			hipFreeAsync(stream_compat->ones.data, stream);
		stream_compat->ones.n = n;
		stream_compat->ones.data = (float*)cumalloc(device, sizeof(float) * n);
		const int block_x = (n + 255) >> 8;
		ones<<<block_x, 256, 0, stream>>>(stream_compat->ones.data, n);
	}
	return stream_compat->ones.data;
}

#ifdef HAVE_CUDNN
hipdnnHandle_t ccv_nnc_stream_context_get_cudnn(const ccv_nnc_stream_context_t* const stream_context)
{
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = &ccv_nnc_per_thread_gpu_stream_context;
	if (!stream_compat->cudnn)
	{
		int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
		hipSetDevice(device);
		CUDNN_ENFORCE(hipdnnCreate(&stream_compat->cudnn));
		CUDNN_ENFORCE(hipdnnSetStream(stream_compat->cudnn, stream_compat->stream));
	}
	return stream_compat->cudnn;
}

hipdnnActivationDescriptor_t ccv_nnc_stream_context_get_activation_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnActivationDescriptor_t desc;
	hipdnnCreateActivationDescriptor(&desc);
	return desc;
}

hipdnnConvolutionDescriptor_t ccv_nnc_stream_context_get_convolution_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnConvolutionDescriptor_t desc;
	hipdnnCreateConvolutionDescriptor(&desc);
	return desc;
}

hipdnnDropoutDescriptor_t ccv_nnc_stream_context_get_dropout_descriptor(const ccv_nnc_stream_context_t* const stream_context, const float p)
{
	hipdnnDropoutDescriptor_t desc;
	hipdnnCreateDropoutDescriptor(&desc);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	ccv_nnc_stream_context_compat_t* stream_compat = (ccv_nnc_stream_context_compat_t*)stream_context;
	if (!stream_compat)
		stream_compat = &ccv_nnc_per_thread_gpu_stream_context;
	size_t state_size;
	hipdnnDropoutGetStatesSize(cudnn, &state_size);
	if (stream_compat->rngs)
	{
#if CUDNN_VERSION >= 7100
		cudnnRestoreDropoutDescriptor(desc, cudnn, p, stream_compat->rngs, state_size, stream_compat->seed);
#else
		++stream_compat->seed;
		hipdnnSetDropoutDescriptor(desc, cudnn, p, stream_compat->rngs, state_size, stream_compat->seed);
#endif
	} else {
		int device = CCV_STREAM_GET_DEVICE_ID(stream_compat->type);
		hipSetDevice(device);
		hipMalloc(&stream_compat->rngs, state_size);
		stream_compat->seed = (unsigned long long)stream_compat;
		hipdnnSetDropoutDescriptor(desc, cudnn, p, stream_compat->rngs, state_size, stream_compat->seed);
	}
	return desc;
}

hipdnnFilterDescriptor_t ccv_nnc_stream_context_get_filter_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnFilterDescriptor_t desc;
	hipdnnCreateFilterDescriptor(&desc);
	return desc;
}

hipdnnOpTensorDescriptor_t ccv_nnc_stream_context_get_op_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnOpTensorDescriptor_t desc;
	hipdnnCreateOpTensorDescriptor(&desc);
	return desc;
}

hipdnnPoolingDescriptor_t ccv_nnc_stream_context_get_pooling_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnPoolingDescriptor_t desc;
	hipdnnCreatePoolingDescriptor(&desc);
	return desc;
}

hipdnnReduceTensorDescriptor_t ccv_nnc_stream_context_get_reduce_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnReduceTensorDescriptor_t desc;
	hipdnnCreateReduceTensorDescriptor(&desc);
	return desc;
}

hipdnnTensorDescriptor_t ccv_nnc_stream_context_get_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context)
{
	hipdnnTensorDescriptor_t desc;
	hipdnnCreateTensorDescriptor(&desc);
	return desc;
}

void ccv_nnc_stream_context_return_activation_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnActivationDescriptor_t activation_desc)
{
	hipdnnDestroyActivationDescriptor(activation_desc);
}

void ccv_nnc_stream_context_return_convolution_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnConvolutionDescriptor_t convolution_desc)
{
	hipdnnDestroyConvolutionDescriptor(convolution_desc);
}

void ccv_nnc_stream_context_return_dropout_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnDropoutDescriptor_t dropout_desc)
{
	hipdnnDestroyDropoutDescriptor(dropout_desc);
}

void ccv_nnc_stream_context_return_filter_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnFilterDescriptor_t filter_desc)
{
	hipdnnDestroyFilterDescriptor(filter_desc);
}

void ccv_nnc_stream_context_return_op_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnOpTensorDescriptor_t op_tensor_desc)
{
	hipdnnDestroyOpTensorDescriptor(op_tensor_desc);
}

void ccv_nnc_stream_context_return_pooling_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnPoolingDescriptor_t pooling_desc)
{
	hipdnnDestroyPoolingDescriptor(pooling_desc);
}

void ccv_nnc_stream_context_return_reduce_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnReduceTensorDescriptor_t reduce_tensor_desc)
{
	hipdnnDestroyReduceTensorDescriptor(reduce_tensor_desc);
}

void ccv_nnc_stream_context_return_tensor_descriptor(const ccv_nnc_stream_context_t* const stream_context, hipdnnTensorDescriptor_t tensor_desc)
{
	hipdnnDestroyTensorDescriptor(tensor_desc);
}

ccv_nnc_cudnn_tensor_view_descriptor_t ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(const ccv_nnc_stream_context_t* const stream_context, const ccv_nnc_tensor_view_t* const tensor)
{
	ccv_nnc_cudnn_tensor_view_descriptor_t tensor_desc = {
		stream_context,
		ccv_nnc_stream_context_get_tensor_descriptor(stream_context),
		tensor->data,
	};
	int dim[CCV_NNC_MAX_DIM_ALLOC] = {};
	int stride[CCV_NNC_MAX_DIM_ALLOC] = {};
	const int axis_count = ccv_nnc_tensor_nd(tensor->info.dim);
	const int* const inc = CCV_IS_TENSOR_VIEW(tensor) ? tensor->inc : tensor->info.dim;
	int i;
	const int offset = CCV_NNC_MAX_DIM + 2 - axis_count;
	if (offset > 0)
	{
		dim[CCV_NNC_MAX_DIM + 1] = tensor->info.dim[axis_count - 1];
		stride[CCV_NNC_MAX_DIM + 1] = 1;
		for (i = CCV_NNC_MAX_DIM; i >= offset; i--)
		{
			dim[i] = tensor->info.dim[i - offset];
			stride[i] = stride[i + 1] * inc[i - offset + 1];
		}
		for (i = 0; i < offset; i++)
		{
			dim[i] = 1;
			stride[i] = stride[offset];
		}
	} else {
		dim[axis_count - 1] = tensor->info.dim[axis_count - 1];
		stride[axis_count - 1] = 1;
		for (i = axis_count - 2; i >= 0; i--)
		{
			dim[i] = tensor->info.dim[i];
			stride[i] = stride[i + 1] * inc[i + 1];
		}
	}
	if (axis_count <= 4)
	{
		CUDNN_ENFORCE(hipdnnSetTensor4dDescriptorEx(tensor_desc.descriptor, HIPDNN_DATA_FLOAT, dim[0], dim[1], dim[2], dim[3], stride[0], stride[1], stride[2], stride[3]));
	} else {
		CUDNN_ENFORCE(hipdnnSetTensorNdDescriptor(tensor_desc.descriptor, HIPDNN_DATA_FLOAT, axis_count, dim, stride));
	}
	return tensor_desc;
}

ccv_nnc_cudnn_tensor_view_descriptor_t ccv_nnc_cudnn_get_tensor_view_descriptor(const ccv_nnc_stream_context_t* const stream_context, const ccv_nnc_tensor_view_t* const tensor)
{
	ccv_nnc_cudnn_tensor_view_descriptor_t tensor_desc = {
		stream_context,
		ccv_nnc_stream_context_get_tensor_descriptor(stream_context),
		tensor->data,
	};
	// Fill up dimensions with 1s.
	int dim[CCV_NNC_MAX_DIM_ALLOC] = {};
	int stride[CCV_NNC_MAX_DIM_ALLOC] = {};
	const int axis_count = ccv_nnc_tensor_nd(tensor->info.dim);
	const int* const inc = CCV_IS_TENSOR_VIEW(tensor) ? tensor->inc : tensor->info.dim;
	int i;
	if (tensor->info.format == CCV_TENSOR_FORMAT_NCHW)
	{
		switch (axis_count)
		{
			case 1:
				dim[0] = dim[2] = dim[3] = 1;
				dim[1] = tensor->info.dim[0];
				stride[0] = inc[0];
				stride[1] = 1;
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = 1;
				break;
			case 2:
				dim[0] = tensor->info.dim[0];
				dim[1] = tensor->info.dim[1];
				dim[2] = dim[3] = 1;
				stride[0] = inc[1];
				stride[1] = 1;
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = 1;
				break;
			case CCV_NNC_MAX_DIM + 1:
				dim[0] = 1;
				dim[1] = tensor->info.dim[0];
				stride[CCV_NNC_MAX_DIM + 1] = 1;
				for (i = CCV_NNC_MAX_DIM - 1; i >= 0; i--)
				{
					dim[i + 2] = tensor->info.dim[i + 1];
					stride[i + 1] = stride[i + 2] * inc[i + 1];
				}
				stride[0] = stride[1] * inc[0];
				break;
			case CCV_NNC_MAX_DIM + 2:
				stride[CCV_NNC_MAX_DIM + 1] = 1;
				dim[CCV_NNC_MAX_DIM + 1] = tensor->info.dim[CCV_NNC_MAX_DIM + 1];
				for (i = CCV_NNC_MAX_DIM; i >= 0; i--)
				{
					dim[i] = tensor->info.dim[i];
					stride[i] = stride[i + 1] * inc[i + 1];
				}
				break;
			default:
				assert(0);
		}
	} else if (tensor->info.format == CCV_TENSOR_FORMAT_NHWC) {
		switch (axis_count)
		{
			case 1:
				dim[0] = dim[2] = dim[3] = 1;
				dim[1] = tensor->info.dim[0];
				stride[0] = inc[0];
				stride[1] = 1;
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = 1; // Even though technically this should be inc[1] (because hw is after c), however, make it 1 doesn't have any differences and more versatile.
				break;
			case 2:
				dim[0] = tensor->info.dim[0];
				dim[1] = tensor->info.dim[1];
				dim[2] = dim[3] = 1;
				stride[0] = inc[1];
				stride[1] = 1;
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = 1; // Even though technically this should be inc[1] (because hw is after c), however, make it 1 doesn't have any differences and more versatile.
				break;
			case CCV_NNC_MAX_DIM + 1:
				dim[0] = 1;
				dim[1] = tensor->info.dim[CCV_NNC_MAX_DIM];
				stride[1] = 1;
				for (i = CCV_NNC_MAX_DIM - 1; i >= 0; i--)
				{
					dim[i + 2] = tensor->info.dim[i];
					stride[i + 2] = (i == CCV_NNC_MAX_DIM - 1) ? inc[i + 1] : stride[i + 3] * inc[i + 1];
				}
				stride[0] = stride[2] * inc[0];
				break;
			case CCV_NNC_MAX_DIM + 2:
				dim[0] = tensor->info.dim[0];
				dim[1] = tensor->info.dim[CCV_NNC_MAX_DIM + 1];
				stride[1] = 1;
				for (i = CCV_NNC_MAX_DIM - 1; i >= 0; i--)
				{
					dim[i + 2] = tensor->info.dim[i + 1];
					stride[i + 2] = (i == CCV_NNC_MAX_DIM - 1) ? inc[i + 2] : stride[i + 3] * inc[i + 2];
				}
				stride[0] = stride[2] * inc[1];
				break;
			default:
				assert(0);
		}
	} else if (tensor->info.format == CCV_TENSOR_FORMAT_CHWN) {
		switch (axis_count)
		{
			case 1:
				dim[0] = dim[2] = dim[3] = 1;
				dim[1] = tensor->info.dim[0];
				stride[0] = inc[0];
				stride[1] = 1;
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = 1;
				break;
			case 2:
				dim[0] = tensor->info.dim[1];
				dim[1] = tensor->info.dim[0];
				dim[2] = dim[3] = 1;
				stride[0] = 1;
				stride[1] = inc[1];
				for (i = 2; i < CCV_NNC_MAX_DIM + 2; i++)
					stride[i] = inc[1];
				break;
			case CCV_NNC_MAX_DIM + 1:
				dim[0] = 1;
				dim[1] = tensor->info.dim[0];
				stride[CCV_NNC_MAX_DIM + 1] = 1;
				for (i = CCV_NNC_MAX_DIM - 1; i >= 0; i--)
				{
					dim[i + 2] = tensor->info.dim[i + 1];
					stride[i + 1] = stride[i + 2] * inc[i + 1];
				}
				stride[0] = stride[1] * inc[0];
				break;
			case CCV_NNC_MAX_DIM + 2:
				dim[0] = tensor->info.dim[CCV_NNC_MAX_DIM + 1];
				stride[0] = 1;
				dim[CCV_NNC_MAX_DIM + 1] = tensor->info.dim[CCV_NNC_MAX_DIM];
				stride[CCV_NNC_MAX_DIM + 1] = inc[CCV_NNC_MAX_DIM + 1];
				for (i = CCV_NNC_MAX_DIM; i > 0; i--)
				{
					dim[i] = tensor->info.dim[i - 1];
					stride[i] = stride[i + 1] * inc[i]; // inc[i] is actually the one before.
				}
				break;
			default:
				assert(0);
		}
	}
	if (CCV_NNC_MAX_DIM == 2)
	{
		CUDNN_ENFORCE(hipdnnSetTensor4dDescriptorEx(tensor_desc.descriptor, HIPDNN_DATA_FLOAT, dim[0], dim[1], dim[2], dim[3], stride[0], stride[1], stride[2], stride[3]));
	} else {
		CUDNN_ENFORCE(hipdnnSetTensorNdDescriptor(tensor_desc.descriptor, HIPDNN_DATA_FLOAT, CCV_NNC_MAX_DIM + 2, dim, stride));
	}
	return tensor_desc;
}

void ccv_nnc_cudnn_deinit_tensor_view_descriptor(const ccv_nnc_cudnn_tensor_view_descriptor_t tensor_desc)
{
	ccv_nnc_stream_context_return_tensor_descriptor(tensor_desc.stream_context, tensor_desc.descriptor);
}

ccv_nnc_cudnn_filter_descriptor_t ccv_nnc_cudnn_get_filter_descriptor(const ccv_nnc_stream_context_t* const stream_context, const ccv_nnc_tensor_t* const tensor)
{
	ccv_nnc_cudnn_filter_descriptor_t filter_desc = {
		stream_context,
		ccv_nnc_stream_context_get_filter_descriptor(stream_context),
		tensor->data,
	};
	assert(!CCV_IS_TENSOR_VIEW(tensor));
	const int nd = ccv_nnc_tensor_nd(tensor->info.dim);
	assert(nd == CCV_NNC_MAX_DIM + 2);
	int dim[CCV_NNC_MAX_DIM_ALLOC] = {};
	int i;
	if (tensor->info.format == CCV_TENSOR_FORMAT_NCHW)
	{
		for (i = 0; i < nd; i++)
			dim[i] = tensor->info.dim[i];
		if (nd == 4)
		{
			CUDNN_ENFORCE(hipdnnSetFilter4dDescriptor(filter_desc.descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, dim[0], dim[1], dim[2], dim[3]));
		} else {
			CUDNN_ENFORCE(hipdnnSetFilterNdDescriptor(filter_desc.descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, nd, dim));
		}
	} else if (tensor->info.format == CCV_TENSOR_FORMAT_NHWC) {
		dim[0] = tensor->info.dim[0];
		dim[1] = tensor->info.dim[nd - 1];
		for (i = 2; i < nd; i++)
			dim[i] = tensor->info.dim[i - 1];
		if (nd == 4)
		{
			CUDNN_ENFORCE(hipdnnSetFilter4dDescriptor(filter_desc.descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NHWC, dim[0], dim[1], dim[2], dim[3]));
		} else {
			CUDNN_ENFORCE(hipdnnSetFilterNdDescriptor(filter_desc.descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NHWC, nd, dim));
		}
	}
	return filter_desc;
}

void ccv_nnc_cudnn_deinit_filter_descriptor(const ccv_nnc_cudnn_filter_descriptor_t filter_desc)
{
	ccv_nnc_stream_context_return_filter_descriptor(filter_desc.stream_context, filter_desc.descriptor);
}

ccv_nnc_cudnn_convolution_descriptor_t ccv_nnc_cudnn_get_convolution_descriptor(const ccv_nnc_stream_context_t* const stream_context, const ccv_nnc_hint_t hint)
{
	ccv_nnc_cudnn_convolution_descriptor_t convolution_desc = {
		stream_context,
		ccv_nnc_stream_context_get_convolution_descriptor(stream_context),
	};
	int i;
	int p[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		p[i] = ccv_max(hint.border.begin[i], hint.border.end[i]);
	int v[CCV_NNC_MAX_DIM];
	for (i = 0; i < CCV_NNC_MAX_DIM; i++)
		v[i] = hint.stride.dim[i];
	if (CCV_NNC_MAX_DIM == 2)
	{
		CUDNN_ENFORCE(hipdnnSetConvolution2dDescriptor(convolution_desc.descriptor, p[0], p[1], v[0], v[1], 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
	} else {
		int u[CCV_NNC_MAX_DIM];
		for (i = 0; i < CCV_NNC_MAX_DIM; i++)
			u[i] = 1;
		CUDNN_ENFORCE(hipdnnSetConvolutionNdDescriptor(convolution_desc.descriptor, CCV_NNC_MAX_DIM, p, v, u, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
	}
	return convolution_desc;
}

void ccv_nnc_cudnn_deinit_convolution_descriptor(const ccv_nnc_cudnn_convolution_descriptor_t convolution_desc)
{
	ccv_nnc_stream_context_return_convolution_descriptor(convolution_desc.stream_context, convolution_desc.descriptor);
}
#endif

static void _ccv_nnc_cufree_stream_callback(hipStream_t stream, hipError_t status, void* ptr)
{
	hipFree(ptr);
}

void hipFreeAsync(void* ptr, hipStream_t stream)
{
	hipStreamAddCallback(stream, _ccv_nnc_cufree_stream_callback, ptr, 0);
}
